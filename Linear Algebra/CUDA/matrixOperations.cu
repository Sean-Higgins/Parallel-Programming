#include "hip/hip_runtime.h"
// System Includes
#include <stdio.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>
#include <assert.h>
#include <malloc.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

// CudaCheckError: Prints a detailed message abount the most recent
//                 error to have occurred.
void CudaCheckError(int which) {
    // "which" is which error number has been assigned:
    // It is not a line number, just an arbitrary integer
    hipError_t e = hipGetLastError();

    if (e != hipSuccess)
        fprintf(stderr, "CUDA Failure: %s:%d: '%s'\n", __FILE__, which, hipGetErrorString(e));
}


#define IN
#define OUT

__global__ void MatrixMult(IN float *dMatrixA, IN float *dMatrixB, IN int *dMW, OUT float *dMatrixC) {
    //unsigned int numItems = blockDim.x;
    //unsigned int wgNum    = blockIdx.x;   // Work-Group Number
    //unsigned int tnum     = threadIdx.x;
    unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

    dMatrixC[gid] = 0.0;

    // 
}

int main(int argc, char *argv[]) {
    
}